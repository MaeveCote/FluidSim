#include "hip/hip_runtime.h"
#include "exercises/VectorAdditionKernel.cuh"

#include <iostream>

using namespace std;

namespace exercises
{
  __global__ void VecAddKernel(const float* a, const float* b, float* c, int n)
  {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    while (tid < n)
    {
      c[tid] = a[tid] + b[tid];
      tid += blockDim.x * gridDim.x;
    }
  }

  void VectorAdder::Add(const float* a, const float* b, float* c, int n)
  {
    float *dev_a, *dev_b, *dev_c;

    // Allocate memory on the device
    CUDA_CHECK(hipMalloc((void**)&dev_a, sizeof(float) * n));
    CUDA_CHECK(hipMalloc((void**)&dev_b, sizeof(float) * n));
    CUDA_CHECK(hipMalloc((void**)&dev_c, sizeof(float) * n));

    // Copy memory to device
    CUDA_CHECK(hipMemcpy(dev_a, a, sizeof(float) * n, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dev_b, b, sizeof(float) * n, hipMemcpyHostToDevice));
    
    // Setup record process time
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start));

    // Launch the kernel
    int blocks = (n + ThreadsPerBlock - 1) / ThreadsPerBlock;
    VecAddKernel<<<blocks, ThreadsPerBlock>>>(dev_a, dev_b, dev_c, n);
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    // Find the process time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "Kernel execution time: " << milliseconds << " ms" << endl;

    // Copy back to host and cleanup
    CUDA_CHECK(hipMemcpy(c, dev_c, sizeof(float) * n, hipMemcpyDeviceToHost));

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    cout << "Finished adding vectors." << endl;
  }
}
